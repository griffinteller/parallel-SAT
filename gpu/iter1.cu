
#include <hip/hip_runtime.h>
#include <cassert>
#include <chrono>
#include <functional>
#include <iostream>
#include <memory>
#include <optional>
#include <vector>
#include <fstream>
#include <sstream>

#define CC(x) { \
    hipError_t err = (x); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error (" << __LINE__ << "): " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

struct Clause {
    int count;
    int* literals;
};

struct Formula {
    int numLiterals;
    int numClauses;
    Clause* clauses;
};

enum AssignedValue {
    UNASSIGNED = 0,
    TRUE,
    FALSE
};

struct Assignment {
    AssignedValue* values;
};

__constant__ Formula devFormula;

__device__ AssignedValue flipTF(AssignedValue value) {
    return value == TRUE ? FALSE : value == FALSE ? TRUE : UNASSIGNED;
}

__device__ AssignedValue getAssig(const volatile Assignment* assignment, int literal) {
    if (literal > 0) {
        return assignment->values[literal - 1];
    } else {
        return flipTF(assignment->values[-literal - 1]);
    }
}

__device__ void setAssig(volatile Assignment* assignment, int literal, AssignedValue value) {
    if (literal > 0) {
        assignment->values[literal - 1] = value;
    } else {
        assignment->values[-literal - 1] = flipTF(value);
    }
}


// any propogated must be cleared to false
__global__ void propagateUnits(volatile Assignment* assignment, volatile bool* anyPropagated) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= devFormula.numClauses) return;

    const Clause& clause = devFormula.clauses[tid];

    int numUnassigned = 0;
    int lastUnassigned;
    for (int i = 0; i < clause.count; i++) {
        int literal = clause.literals[i];
        AssignedValue value = getAssig(assignment, literal);
        if (value == TRUE) {
            // clause is satisfied
            return;
        }
        if (value == UNASSIGNED) {
            numUnassigned++;
            lastUnassigned = clause.literals[i];
        }
        if (numUnassigned > 1) return;
    }

    if (numUnassigned != 1) return;
    setAssig(assignment, lastUnassigned, TRUE);
    *anyPropagated = true;
}

__global__ void propagatePure(volatile Assignment* assignment) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= devFormula.numLiterals) return;

    AssignedValue value = assignment->values[tid];
    if (value != UNASSIGNED) return;

    bool foundAny = false;
    bool polarity;
    for (int i = 0; i < devFormula.numClauses; i++) {
        const Clause& clause = devFormula.clauses[i];
        for (int j = 0; j < clause.count; j++) {
            if (abs(clause.literals[j]) - 1 == tid) {
                bool currPolarity = clause.literals[j] - 1 == tid;
                if (foundAny) {
                    if (polarity != currPolarity) {
                        return;
                    }
                } else {
                    foundAny = true;
                    polarity = currPolarity;
                }
            }
        }
    }

    // we are a pure literal
    setAssig(assignment, tid + 1, polarity ? TRUE : FALSE);
}

__global__ void checkClauses(const volatile Assignment* assignment, volatile bool* anyFalseClauses, volatile bool* allClausesSatisfied) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= devFormula.numClauses) return;

    Clause clause = devFormula.clauses[tid];
    bool clauseSatisfied = false;
    bool clauseFalse = true;
    for (int i = 0; i < clause.count; i++) {
        int literal = clause.literals[i];
        AssignedValue value = getAssig(assignment, literal);

        if (value == TRUE) {
            clauseSatisfied = true;
            clauseFalse = false;
            break;
        }

        if (value == UNASSIGNED) {
            clauseFalse = false;
            break;
        }
    }

    if (clauseFalse) *anyFalseClauses = true;
    if (!clauseSatisfied) *allClausesSatisfied = false;
}

void copyFormulaToDevice(const Formula& formula) {
    Clause* clauses;
    CC(hipMalloc(&clauses, sizeof(Clause) * formula.numClauses));
    for (int i = 0; i < formula.numClauses; i++) {
        CC(hipMemcpy(&clauses[i].count, &formula.clauses[i].count, sizeof(int), hipMemcpyHostToDevice));

        int* literals;
        CC(hipMalloc(&literals, sizeof(int) * formula.clauses[i].count));
        CC(hipMemcpy(literals, formula.clauses[i].literals, sizeof(int) * formula.clauses[i].count, hipMemcpyHostToDevice));
        CC(hipMemcpy(&clauses[i].literals, &literals, sizeof(int*), hipMemcpyHostToDevice));
    }

    Formula copy = {
        .numLiterals = formula.numLiterals,
        .numClauses = formula.numClauses,
        .clauses = clauses
    };
    CC(hipMemcpyToSymbol(HIP_SYMBOL(devFormula), &copy, sizeof(Formula)));
}

bool dpllHostDirected(const Formula& formula) {
    const auto startTime = std::chrono::high_resolution_clock::now();

    std::cout << "dpllHostDirected" << std::endl;
    std::cout << "numLiterals: " << formula.numLiterals << std::endl;
    std::cout << "numClauses: " << formula.numClauses << std::endl;

    copyFormulaToDevice(formula);

    Assignment devAssignmentView;
    CC(hipMalloc(&devAssignmentView.values, sizeof(int) * formula.numLiterals));

    std::vector<AssignedValue> literalZeros(formula.numLiterals, UNASSIGNED);
    CC(hipMemcpy(devAssignmentView.values, literalZeros.data(), sizeof(int) * formula.numLiterals, hipMemcpyHostToDevice));

    Assignment* devAssignment;
    CC(hipMalloc(&devAssignment, sizeof(Assignment)));
    CC(hipMemcpy(devAssignment, &devAssignmentView, sizeof(Assignment), hipMemcpyHostToDevice));

    bool* anyUnitPropagated;
    bool* allClausesSatisfied;
    bool* anyFalseClauses;
    CC(hipMalloc(&allClausesSatisfied, sizeof(bool)));
    CC(hipMalloc(&anyFalseClauses, sizeof(bool)));
    CC(hipMalloc(&anyUnitPropagated, sizeof(bool)));

    std::function<bool()> inner = [=, &inner, &formula] () -> bool {
        bool f = false;
        bool t = true;

        constexpr int clauseBlockSize = 256;
        int numClauseBlocks = (formula.numClauses - 1) / clauseBlockSize + 1;

        constexpr int litBlockSize = 256;
        int numLitBlocks = (formula.numLiterals - 1) / litBlockSize + 1;

        // unit literal propagation
        while (true) {
            CC(hipMemcpy(anyUnitPropagated, &f, sizeof(bool), hipMemcpyHostToDevice));

            propagateUnits<<<numClauseBlocks, clauseBlockSize>>>(devAssignment, anyUnitPropagated);

            bool shouldContinue;
            CC(hipMemcpy(&shouldContinue, anyUnitPropagated, sizeof(bool), hipMemcpyDeviceToHost));

            // std::cout << "shouldContinue: " << shouldContinue << std::endl;
            if (!shouldContinue) break;
        }
    

        // copy and print assignment
        std::vector<AssignedValue> retAssignment(formula.numLiterals);
        hipMemcpy(retAssignment.data(), devAssignmentView.values, sizeof(int) * formula.numLiterals, hipMemcpyDeviceToHost);

        CC(hipMemcpy(allClausesSatisfied, &t, sizeof(bool), hipMemcpyHostToDevice));
        CC(hipMemcpy(anyFalseClauses, &f, sizeof(bool), hipMemcpyHostToDevice));
        checkClauses<<<numClauseBlocks, clauseBlockSize>>>(devAssignment, anyFalseClauses, allClausesSatisfied);

        bool allTrue;
        CC(hipMemcpy(&allTrue, allClausesSatisfied, sizeof(bool), hipMemcpyDeviceToHost));
        if (allTrue) {
            return true;
        }

        bool anyFalse;
        CC(hipMemcpy(&anyFalse, anyFalseClauses, sizeof(bool), hipMemcpyDeviceToHost));
        if (anyFalse) {
            return false;
        }

        // find unassigned literal
        int unassignedLiteral = 0;
        for (int i = 0; i < formula.numLiterals; i++) {
            if (retAssignment[i] == UNASSIGNED) {
                unassignedLiteral = i + 1;
                break;
            }
        }

        assert(unassignedLiteral > 0);

        retAssignment[unassignedLiteral - 1] = TRUE;
        CC(hipMemcpy(devAssignmentView.values, retAssignment.data(), sizeof(int) * formula.numLiterals, hipMemcpyHostToDevice));
        if (inner()) return true;

        retAssignment[unassignedLiteral - 1] = FALSE;
        CC(hipMemcpy(devAssignmentView.values, retAssignment.data(), sizeof(int) * formula.numLiterals, hipMemcpyHostToDevice));
        return inner();
    };

    const auto initializationEndTime = std::chrono::high_resolution_clock::now();
    const auto initializationDuration = std::chrono::duration_cast<std::chrono::milliseconds>(initializationEndTime - startTime);
    std::cout << "Initialization time: " << initializationDuration.count() << " ms" << std::endl;

   bool res = inner();

    const auto endTime = std::chrono::high_resolution_clock::now();
    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - initializationEndTime);
    std::cout << "Total time: " << duration.count() << " ms" << std::endl;

    return res;
}

int main(int argc, char** argv) {
    // usage: solver [-P] <benchmark_file_path>
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <benchmark_file_path>" << std::endl;
        return 1;
    }

    std::string file_path = argv[1];
    std::ifstream infile(file_path);
    if (!infile) {
        std::cerr << "Error: Unable to open file " << file_path << std::endl;
        return 1;
    }

    // parse CNF file to generate formula
    std::vector<Clause> clauses;
    std::string line;
    bool pLineFound = false;
    int numLiterals;
    while (std::getline(infile, line)) {
        if (line.empty() || line[0] == 'c') {
            continue;
        }
        if (line[0] == 'p') {
            pLineFound = true;
            // parse the problem line
            std::istringstream iss(line);
            std::string tmp;
            iss >> tmp >> tmp >> numLiterals;
            continue;
        }
        if (pLineFound) {
            std::istringstream iss(line);
            int lit;
            std::vector<int> clause;
            while (iss >> lit) {
                if (lit == 0) {
                    break;
                }
                clause.push_back(lit);
            }
            if (!clause.empty()) {
                Clause c;
                c.count = clause.size();
                c.literals = new int[c.count];
                for (int i = 0; i < c.count; i++) {
                    c.literals[i] = clause[i];
                }
                clauses.push_back(c);
            }
        }
    }
    infile.close();

    Formula formula;
    formula.numLiterals = numLiterals;
    formula.numClauses = clauses.size();
    formula.clauses = new Clause[formula.numClauses];
    for (size_t i = 0; i < clauses.size(); i++) {
        formula.clauses[i].count = clauses[i].count;
        formula.clauses[i].literals = clauses[i].literals;
    }

    bool sat = dpllHostDirected(formula);
    if (sat) {
        std::cout << "SAT" << std::endl;
    } else {
        std::cout << "UNSAT" << std::endl;
    }

    return 0;
}
